#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <ctime>
#include <chrono>
#include <random>
#include <omp.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <>

using namespace std;

const int POPULATION_SIZE = 500;
const int GENERATIONS = 1000;
const int BEST_COUNT = 50;
const double MUTATION_RATE = 0.2;
const int BOARD_SIZE = 9;
const int SUBGRID_SIZE = 3;
const int NUM_RUNS = 1;
const int THREADS = 10;

struct Individual {
    int board[BOARD_SIZE][BOARD_SIZE];
    int quality;
};

// Pomocnicza funkcja do drukowania planszy Sudoku
__host__ void print_board(const int board[BOARD_SIZE][BOARD_SIZE]) {
    for (int i = 0; i < BOARD_SIZE; ++i) {
        for (int j = 0; j < BOARD_SIZE; ++j) {
            cout << board[i][j] << " ";
        }
        cout << endl;
    }
}

// Funkcja zwracająca dostępne liczby dla danego pola
__device__ void get_possible_numbers(const int grid[BOARD_SIZE][BOARD_SIZE], int row, int col, bool possible_numbers[BOARD_SIZE + 1]) {
    for (int i = 0; i <= BOARD_SIZE; ++i) {
        possible_numbers[i] = true;
    }

    // Sprawdzanie dostępnych liczb w wierszu
    for (int i = 0; i < BOARD_SIZE; ++i) {
        possible_numbers[grid[row][i]] = false;
    }

    // Sprawdzanie dostępnych liczb w kolumnie
    for (int i = 0; i < BOARD_SIZE; ++i) {
        possible_numbers[grid[i][col]] = false;
    }

    // Sprawdzanie dostępnych liczb w kwadracie 3x3
    int start_row = row / SUBGRID_SIZE * SUBGRID_SIZE;
    int start_col = col / SUBGRID_SIZE * SUBGRID_SIZE;
    for (int i = start_row; i < start_row + SUBGRID_SIZE; ++i) {
        for (int j = start_col; j < start_col + SUBGRID_SIZE; ++j) {
            possible_numbers[grid[i][j]] = false;
        }
    }
   
}

// Funkcja do wypełniania pojedynczego Sudoku na podstawie dostępnych liczb
__device__ void shuffle_cell_order(int cell_order[BOARD_SIZE * BOARD_SIZE], hiprandState* state) {
    int remaining = BOARD_SIZE * BOARD_SIZE;
    for (int i = 0; i < BOARD_SIZE * BOARD_SIZE; ++i) {
        int j = hiprand_uniform(state) * remaining; // Losujemy indeks z pozostałych elementów
        int temp = cell_order[i];
        cell_order[i] = cell_order[j];
        cell_order[j] = temp;
        remaining--;
    }
}

__device__ void fill_sudoku(const int initial_board[BOARD_SIZE][BOARD_SIZE], int grid[BOARD_SIZE][BOARD_SIZE], hiprandState* state) {
    for (int i = 0; i < BOARD_SIZE; ++i) {
        for (int j = 0; j < BOARD_SIZE; ++j) {
            grid[i][j] = initial_board[i][j];
        }
    }

    // Losowo ustawiamy kolejność próby wstawienia liczby w każdą komórkę
    int cell_order[BOARD_SIZE * BOARD_SIZE];
    for (int i = 0; i < BOARD_SIZE * BOARD_SIZE; ++i) {
        cell_order[i] = i;
    }
    shuffle_cell_order(cell_order, state);

    for (int k = 0; k < BOARD_SIZE * BOARD_SIZE; ++k) {
        int row = cell_order[k] / BOARD_SIZE;
        int col = cell_order[k] % BOARD_SIZE;

        if (grid[row][col] == 0) {
            bool possible_numbers[BOARD_SIZE + 1];
            get_possible_numbers(grid, row, col, possible_numbers);

            int num_choices = 0;
            for (int i = 1; i <= BOARD_SIZE; ++i) {
                if (possible_numbers[i]) {
                    num_choices++;
                }
            }

            if (num_choices == 0) {
                break; // Przerwij próbę wypełnienia, jeśli brak dostępnych liczb
            }

            int num_index = hiprand(state) % num_choices;
            int num_count = 0;
            for (int i = 1; i <= BOARD_SIZE; ++i) {
                if (possible_numbers[i]) {
                    if (num_count == num_index) {
                        grid[row][col] = i;
                        break;
                    }
                    num_count++;
                }
            }
        }
    }
}

// Funkcja obliczająca ilość pustych pól w planszy
__device__ int count_empty_cells(int board[BOARD_SIZE][BOARD_SIZE]) {
    int empty_cells = 0;
    for (int i = 0; i < BOARD_SIZE; ++i) {
        for (int j = 0; j < BOARD_SIZE; ++j) {
            if (board[i][j] == 0) {
                empty_cells++;
            }
        }
    }
    return empty_cells;
}

__global__ void generate_first_population_kernel(const int initial_board[BOARD_SIZE][BOARD_SIZE], Individual population[POPULATION_SIZE]) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < POPULATION_SIZE) {
        hiprandState state;
        hiprand_init(id, 0, 0, &state);
        fill_sudoku(initial_board, population[id].board, &state); // Wypełnij planszę Sudoku dla nowego osobnika
        population[id].quality = count_empty_cells(population[id].board); // Oblicz jakość planszy dla nowego osobnika
    }
}

void generate_first_population_CUDA(const int initial_board[BOARD_SIZE][BOARD_SIZE], Individual population[POPULATION_SIZE]) {
    // Alokacja pamięci na GPU
    int(*d_initial_board)[BOARD_SIZE];
    hipMalloc((void**)&d_initial_board, BOARD_SIZE * BOARD_SIZE * sizeof(int));
    Individual* d_population;
    hipMalloc((void**)&d_population, POPULATION_SIZE * sizeof(Individual));

    // Kopiowanie danych z hosta do urządzenia
    hipMemcpy(d_initial_board, initial_board, BOARD_SIZE * BOARD_SIZE * sizeof(int), hipMemcpyHostToDevice);

    // Wywołanie kernela
    int threadsPerBlock = THREADS;
    int blocksPerGrid = (POPULATION_SIZE + threadsPerBlock - 1) / threadsPerBlock;
    generate_first_population_kernel << <threadsPerBlock, blocksPerGrid>> > (d_initial_board, d_population);

    // Kopiowanie wyników z urządzenia do hosta
    hipMemcpy(population, d_population, POPULATION_SIZE * sizeof(Individual), hipMemcpyDeviceToHost);

    // Zwolnienie pamięci na GPU
    hipFree(d_initial_board);
    hipFree(d_population);
}

// Funkcja wybierająca najlepsze osobniki z populacji na podstawie jakości planszy
__host__ void select_best_individuals(const Individual population[POPULATION_SIZE], Individual best_individuals[BEST_COUNT]) {
    // Tworzymy kopię populacji, aby nie zmieniać kolejności oryginalnej
    Individual sorted_population[POPULATION_SIZE];
    memcpy(sorted_population, population, sizeof(Individual) * POPULATION_SIZE);

    // Sortujemy populację według jakości planszy (rosnąco)
    sort(sorted_population, sorted_population + POPULATION_SIZE, [](const Individual& a, const Individual& b) {
        return a.quality < b.quality;
        });

    // Wybieramy najlepsze jednostki
    for (int i = 0; i < BEST_COUNT && i < POPULATION_SIZE; ++i) {
        best_individuals[i] = sorted_population[i];
    }
}

__device__ void create_child(const int initial_board[BOARD_SIZE][BOARD_SIZE], const Individual& parent1, const Individual& parent2, Individual& child, hiprandState* state) {
    for (int i = 0; i < BOARD_SIZE; ++i) {
        for (int j = 0; j < BOARD_SIZE; ++j) {
            child.board[i][j] = initial_board[i][j];
        }
    }

    int cell_order[BOARD_SIZE * BOARD_SIZE];
    for (int i = 0; i < BOARD_SIZE * BOARD_SIZE; ++i) {
        cell_order[i] = i;
    }
    shuffle_cell_order(cell_order, state);

    // Losowo przemieszaj kolejność komórek dziecka
    shuffle_cell_order(cell_order, state);

    for (int k = 0; k < BOARD_SIZE * BOARD_SIZE; ++k) {
        int row = cell_order[k] / BOARD_SIZE;
        int col = cell_order[k] % BOARD_SIZE;

        if (child.board[row][col] == 0) {
            // Sprawdzenie mutacji
            if (hiprand_uniform(state) < 0.25) {
                bool possible_numbers[BOARD_SIZE + 1];
                get_possible_numbers(child.board, row, col, possible_numbers);
                int num_choices = 0;
                for (int i = 1; i <= BOARD_SIZE; ++i) {
                    if (possible_numbers[i]) {
                        num_choices++;
                    }
                }
                if (num_choices > 0) {
                    int num_index = hiprand(state) % num_choices;
                    int num_count = 0;
                    for (int i = 1; i <= BOARD_SIZE; ++i) {
                        if (possible_numbers[i]) {
                            if (num_count == num_index) {
                                child.board[row][col] = i;
                                break;
                            }
                            num_count++;
                        }
                    }
                }
            }
            else {
                // Sprawdź dostępność wartości od obu rodziców
                bool possible_numbers[BOARD_SIZE + 1];
                get_possible_numbers(child.board, row, col, possible_numbers);

                // Sprawdź, czy wartość komórki rodzica może wystąpić w komórce dziecka
                bool value_can_be_in_child = possible_numbers[parent1.board[row][col]] || possible_numbers[parent2.board[row][col]];

                // Jeśli wartość może wystąpić w dziecku, przypisz ją
                if (value_can_be_in_child) {
                    if (possible_numbers[parent1.board[row][col]]) {
                        child.board[row][col] = parent1.board[row][col];
                    }
                    else {
                        child.board[row][col] = parent2.board[row][col];
                    }
                }
                else {
                    // W przypadku braku możliwości wystąpienia wartości w dziecku, pozostaw komórkę dziecka pustą
                    child.board[row][col] = 0;
                }
            }
        }
    }
}


// Funkcja tworząca nowe osobniki w populacji na podstawie najlepszych osobników
__global__ void create_new_population_kernel(const int initial_board[BOARD_SIZE][BOARD_SIZE], const Individual best_individuals[BEST_COUNT], Individual population[POPULATION_SIZE], float mutation_rate) {
    int id = threadIdx.x + (blockIdx.x * blockDim.x);

    if (id < POPULATION_SIZE) {
        hiprandState state;
        hiprand_init(id, 0, 0, &state);

        int parent1_idx = hiprand(&state) % BEST_COUNT;
        int parent2_idx = hiprand(&state) % BEST_COUNT;
        create_child(initial_board, best_individuals[parent1_idx], best_individuals[parent2_idx], population[id], &state);

        population[id].quality = count_empty_cells(population[id].board);
    }
}

void create_new_population_CUDA(const int initial_board[BOARD_SIZE][BOARD_SIZE], const Individual best_individuals[BEST_COUNT], Individual population[POPULATION_SIZE], float mutation_rate) {
    // Alokacja pamięci na GPU
    int(*d_initial_board)[BOARD_SIZE];
    hipMalloc((void**)&d_initial_board, BOARD_SIZE * BOARD_SIZE * sizeof(int));
    Individual* d_best_individuals;
    hipMalloc((void**)&d_best_individuals, BEST_COUNT * sizeof(Individual));
    Individual* d_population;
    hipMalloc((void**)&d_population, POPULATION_SIZE * sizeof(Individual));

    // Kopiowanie danych z hosta do urządzenia
    hipMemcpy(d_initial_board, initial_board, BOARD_SIZE * BOARD_SIZE * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_best_individuals, best_individuals, BEST_COUNT * sizeof(Individual), hipMemcpyHostToDevice);

    // Wywołanie kernela
    int threadsPerBlock = THREADS;
    int blocksPerGrid = (POPULATION_SIZE + threadsPerBlock - 1) / threadsPerBlock;
    create_new_population_kernel << <blocksPerGrid, threadsPerBlock >> > (d_initial_board, d_best_individuals, d_population, mutation_rate);

    // Kopiowanie wyników z urządzenia do hosta
    hipMemcpy(population, d_population, POPULATION_SIZE * sizeof(Individual), hipMemcpyDeviceToHost);

    // Zwolnienie pamięci na GPU
    hipFree(d_initial_board);
    hipFree(d_best_individuals);
    hipFree(d_population);
}

// Funkcja sprawdzająca, czy plansza Sudoku spełnia wszystkie reguły gry
bool is_valid_board(int board[BOARD_SIZE][BOARD_SIZE]) {
    // Sprawdzamy wiersze i kolumny
    for (int i = 0; i < BOARD_SIZE; ++i) {
        bool row_nums[BOARD_SIZE + 1] = { false };
        bool col_nums[BOARD_SIZE + 1] = { false };
        for (int j = 0; j < BOARD_SIZE; ++j) {
            // Sprawdzamy wiersz
            if (board[i][j] == 0 || row_nums[board[i][j]]) {
                return false; // Powtórzona liczba w wierszu lub zero
            }
            row_nums[board[i][j]] = true;
            // Sprawdzamy kolumnę
            if (board[j][i] == 0 || col_nums[board[j][i]]) {
                return false; // Powtórzona liczba w kolumnie lub zero
            }
            col_nums[board[j][i]] = true;
        }
    }

    // Sprawdzamy kwadraty 3x3
    for (int i = 0; i < BOARD_SIZE; i += 3) {
        for (int j = 0; j < BOARD_SIZE; j += 3) {
            bool square_nums[BOARD_SIZE + 1] = { false };
            for (int k = i; k < i + 3; ++k) {
                for (int l = j; l < j + 3; ++l) {
                    if (board[k][l] == 0 || square_nums[board[k][l]]) {
                        return false; // Powtórzona liczba w kwadracie 3x3 lub zero
                    }
                    square_nums[board[k][l]] = true;
                }
            }
        }
    }

    return true; // Plansza jest poprawna
}

int main() {
    int initial_board[BOARD_SIZE][BOARD_SIZE] = {
        {8, 0, 0, 0, 0, 0, 1, 4, 9},
        {0, 0, 0, 5, 0, 0, 0, 0, 2},
        {0, 0, 0, 0, 0, 7, 5, 0, 0},
        {0, 0, 0, 9, 0, 0, 7, 0, 5},
        {0, 0, 9, 0, 4, 0, 2, 0, 0},
        {1, 0, 2, 0, 0, 6, 0, 0, 0},
        {0, 0, 4, 6, 0, 0, 0, 0, 0},
        {3, 0, 0, 0, 0, 9, 0, 0, 0},
        {9, 2, 5, 0, 0, 0, 0, 0, 4}
    };

    Individual population[POPULATION_SIZE];
    Individual best_individuals[BEST_COUNT];

    auto start = std::chrono::steady_clock::now();
    // Generowanie początkowej populacji
    generate_first_population_CUDA(initial_board, population);
    
    for (int generation = 0; generation < GENERATIONS; ++generation) {
        // Wybieranie najlepszych osobników
        select_best_individuals(population, best_individuals);
        cout << "Generacja: " << generation << endl;
        print_board(best_individuals[0].board);
        cout << "Liczba pustych miejsc: " << best_individuals[0].quality << endl;
        if (best_individuals[0].quality == 0) {
            break;
        }
        // Tworzenie nowej populacji
        create_new_population_CUDA(initial_board, best_individuals, population, MUTATION_RATE);
    }
    Individual result = best_individuals[0];
    auto end = std::chrono::steady_clock::now();
    cout << "Is final board valid? " << (is_valid_board(result.board) ? "Yes" : "No") << endl;
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
    double duration_seconds = static_cast<double>(duration.count()) / 1000.0; // Konwersja z milisekund na sekundy

    // Teraz możesz wyświetlić czas wykonania
    std::cout << "Time taken: " << duration_seconds << " seconds" << std::endl;

    return 0;
}

